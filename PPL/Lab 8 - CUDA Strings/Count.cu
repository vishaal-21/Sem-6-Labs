#include<stdio.h>
#include<string.h>
#include "hip/hip_runtime.h"


__global__ void countOcc(char *str, char *word, int *count, int len_str, int len_word)
{
    int id=threadIdx.x;

    if(id<len_str-len_word+1)
    {
        int flag=1;
        for(int i=0;i<len_word;i++)
        {
            if(str[id+i]!=word[i])
            {
                flag=0;
                break;
            }
        }

        if(flag)
            atomicAdd(count,1);
    }
}

int main()
{
    int count;

    char str[]="foo la dee da da labs foo da la la foo la labs foo";
    char word[50];
    printf("String is : %s\n",str);
    printf("Enter word to count frequency : ");
    scanf("%s",word);

    int len_str=strlen(str);
    int len_word=strlen(word);

    char *d_word,*d_str;
    int *d_count;

    hipMalloc((void**)&d_word,len_word*sizeof(char));
    hipMalloc((void**)&d_str,len_str*sizeof(char));
    hipMalloc((void**)&d_count,sizeof(int));

    hipMemcpy(d_str,str,len_str*sizeof(char),hipMemcpyHostToDevice);
    hipMemcpy(d_word,word,len_word*sizeof(char),hipMemcpyHostToDevice);

    countOcc<<<1,len_str>>>(d_str,d_word,d_count,len_str,len_word);

    hipMemcpy(&count,d_count,sizeof(int),hipMemcpyDeviceToHost);
    printf("Number of occurances : %d\n",count);

    return 1;
}