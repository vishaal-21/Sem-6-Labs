#include<stdio.h>
#include<string.h>
#include "hip/hip_runtime.h"


__global__ void operate(char *word, char *res, int len, int res_len)
{
    int id=threadIdx.x;
    int temp=len;
    int index=id;

    for(int i=0;i<len;i++)
    {
        if(index<res_len)
        {
            res[index]=word[id];
        }
        index+=temp;
        temp--;
    }
}

int main()
{
    char word[50];
    printf("Enter a word : ");
    scanf("%s",word);

    int len=strlen(word);
    int res_len=(len*(len+1))/2;

    char res[res_len];

    char *d_word,*d_res;

    hipMalloc((void**)&d_word,len*sizeof(char));
    hipMalloc((void**)&d_res,res_len*sizeof(char));

    hipMemcpy(d_word,word,len*sizeof(char),hipMemcpyHostToDevice);

    operate<<<1,len>>>(d_word,d_res,len,res_len);

    hipMemcpy(res,d_res,res_len*sizeof(char),hipMemcpyDeviceToHost);
    res[res_len]='\0';

    printf("Result : %s\n",res);

    return 1;
}